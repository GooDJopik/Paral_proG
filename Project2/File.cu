#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>

#include <string>

using namespace std;

__global__ void matrixMultiplyKernel(int* A, int* B, int* C, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        int sum = 0;
        for (int k = 0; k < size; k++) {
            sum += A[row * size + k] * B[k * size + col];
        }
        C[row * size + col] = sum;
    }
}

vector<vector<int>> generate_random_matrix(int rows, int cols) {
    vector<vector<int>> matrix(rows, vector<int>(cols));
    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<> dis(0, 100);

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            matrix[i][j] = dis(gen);
        }
    }
    return matrix;
}

void write_matrix_to_file(const string& filename, const vector<vector<int>>& matrix) {
    ofstream file(filename);
    if (!file.is_open()) {
        cerr << "Не удалось открыть файл для записи: " << filename << endl;
        return;
    }

    int rows = matrix.size();
    int cols = matrix[0].size();

    file << rows << " " << cols << endl;
    for (const auto& row : matrix) {
        for (int val : row) {
            file << val << " ";
        }
        file << endl;
    }
}

vector<vector<int>> multiply_matrices_cuda(const vector<vector<int>>& A, const vector<vector<int>>& B) {
    int size = A.size();

    int* flatA = new int[size * size];
    int* flatB = new int[size * size];
    int* flatC = new int[size * size];

    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            flatA[i * size + j] = A[i][j];
            flatB[i * size + j] = B[i][j];
        }
    }

    int* d_A, * d_B, * d_C;
    hipError_t err;
    err = hipMalloc(&d_A, size * size * sizeof(int));
    if (err != hipSuccess) {
        cerr << "Ошибка выделения памяти для d_A: " << hipGetErrorString(err) << endl;
        delete[] flatA;
        delete[] flatB;
        delete[] flatC;
        return vector<vector<int>>();
    }

    err = hipMalloc(&d_B, size * size * sizeof(int));
    if (err != hipSuccess) {
        cerr << "Ошибка выделения памяти для d_B: " << hipGetErrorString(err) << endl;
        hipFree(d_A);
        delete[] flatA;
        delete[] flatB;
        delete[] flatC;
        return vector<vector<int>>();
    }

    err = hipMalloc(&d_C, size * size * sizeof(int));
    if (err != hipSuccess) {
        cerr << "Ошибка выделения памяти для d_C: " << hipGetErrorString(err) << endl;
        hipFree(d_A);
        hipFree(d_B);
        delete[] flatA;
        delete[] flatB;
        delete[] flatC;
        return vector<vector<int>>();
    }

    err = hipMemcpy(d_A, flatA, size * size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        cerr << "Ошибка копирования d_A: " << hipGetErrorString(err) << endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        delete[] flatA;
        delete[] flatB;
        delete[] flatC;
        return vector<vector<int>>();
    }

    err = hipMemcpy(d_B, flatB, size * size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        cerr << "Ошибка копирования d_B: " << hipGetErrorString(err) << endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        delete[] flatA;
        delete[] flatB;
        delete[] flatC;
        return vector<vector<int>>();
    }

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((size + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMultiplyKernel <<<blocksPerGrid, threadsPerBlock >>> (d_A, d_B, d_C, size);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        cerr << "Ошибка выполнения ядра: " << hipGetErrorString(err) << endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        delete[] flatA;
        delete[] flatB;
        delete[] flatC;
        return vector<vector<int>>();
    }

    err = hipMemcpy(flatC, d_C, size * size * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        cerr << "Ошибка копирования результата: " << hipGetErrorString(err) << endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        delete[] flatA;
        delete[] flatB;
        delete[] flatC;
        return vector<vector<int>>();
    }

    vector<vector<int>> C(size, vector<int>(size));
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            C[i][j] = flatC[i * size + j];
        }
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] flatA;
    delete[] flatB;
    delete[] flatC;

    return C;
}

int main() {

    setlocale(LC_ALL, "Russian");

    vector<int> sizes = { 100, 200, 300, 400, 500, 1000, 1500, 2000 };

    system("mkdir -p matrices");
    system("mkdir -p results");

    ofstream time_output("results/time_results.txt");
    if (!time_output.is_open()) {
        cerr << "Не удалось открыть файл для записи результатов времени" << endl;
        return 1;
    }

    time_output << "Размер матриц\tВремя выполнения (секунд)\n";

    for (int size : sizes) {
        cout << "Обработка матриц размера " << size << "x" << size << "..." << endl;

        auto A = generate_random_matrix(size, size);
        auto B = generate_random_matrix(size, size);

        string matrixA_file = "matrices/matrixA_" + to_string(size) + ".txt";
        string matrixB_file = "matrices/matrixB_" + to_string(size) + ".txt";
        write_matrix_to_file(matrixA_file, A);
        write_matrix_to_file(matrixB_file, B);

        auto start = chrono::high_resolution_clock::now();
        vector<vector<int>> C = multiply_matrices_cuda(A, B);
        auto end = chrono::high_resolution_clock::now();

        if (C.empty()) {
            cerr << "Ошибка при умножении матриц размера " << size << "x" << size << endl;
            continue;
        }

        string resultFile = "results/result_" + to_string(size) + ".txt";
        write_matrix_to_file(resultFile, C);

        chrono::duration<double> duration = end - start;
        time_output << size << "x" << size << "\t" << duration.count() << endl;
        cout << "Завершено за " << duration.count() << " секунд" << endl;
    }

    time_output.close();
    cout << "Все задачи выполнены" << endl;

    return 0;
}